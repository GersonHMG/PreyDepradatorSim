#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include "config.hpp"
#include "world_aos.cuh"
#include "world_shared.cuh"
#include "world_cpu.cuh"

void renderGrid(sf::RenderWindow* window, int size_x, int size_y){
    int num_lines = size_y + size_x - 2;
    sf::VertexArray grid(sf::Lines, 2*(num_lines));
    window->setView( window->getDefaultView() );
    auto size = window->getView().getSize();
    float rowH = size.y/size_y;
    float colW = size.x/size_x;
    // row separators
    for(int i=0; i < size_y-1; i++){
        int r = i+1;
        float rowY = rowH*r;
        grid[i*2].position = {0, rowY};
        grid[i*2+1].position = {size.x, rowY};
    }
    // column separators
    for(int i=size_y-1; i < num_lines; i++){
        int c = i-size_y+2;
        float colX = colW*c;
        grid[i*2].position = {colX, 0};
        grid[i*2+1].position = {colX, size.y};  
    }
    for (int i = 0; i < 2*(num_lines); i++){ // Color
        grid[i].color = sf::Color(169,169,169);
    } 
    window->draw(grid);
}

void renderPrey(sf::RenderWindow* window, int x, int y){
    sf::CircleShape sprite(50, 4);
    sprite.rotate(45);
    sprite.scale(sf::Vector2f(0.1f,0.1f));
    sf::Color prey_color = sf::Color::Green;
    sprite.setFillColor( prey_color );
    // Local to global
    float center = (SCREEN_WIDTH/WORLD_DIMENSION)*0.5;
    float x_f = (SCREEN_WIDTH/WORLD_DIMENSION)*x + center;
    float y_f = (SCREEN_HEIGHT/WORLD_DIMENSION)*y + center;
    sprite.setPosition( sf::Vector2f( x_f, y_f ) );
    window->draw(sprite);
}

void renderPredator(sf::RenderWindow* window, int x, int y){
    sf::CircleShape sprite(50, 4);
    sprite.rotate(45);
    sprite.scale(sf::Vector2f(0.1f,0.1f));
    sf::Color prey_color = sf::Color::Red;
    sprite.setFillColor( prey_color );
    // Local to global
    float center = (SCREEN_WIDTH/WORLD_DIMENSION)*0.5;
    float x_f = (SCREEN_WIDTH/WORLD_DIMENSION)*x + center;
    float y_f = (SCREEN_HEIGHT/WORLD_DIMENSION)*y + center;
    sprite.setPosition( sf::Vector2f( x_f, y_f ) );
    window->draw(sprite);
}

void render(Agent* world, sf::RenderWindow* window){
    int size = WORLD_DIMENSION*WORLD_DIMENSION;
    renderGrid(window, WORLD_DIMENSION,WORLD_DIMENSION);

    for (int i = 0; i < size; i++){
        switch (world[i].cell_type){
        case PREY:
            renderPrey(window, i%WORLD_DIMENSION , i/WORLD_DIMENSION );
            break;
        case PREDATOR:
            renderPredator(window, i%WORLD_DIMENSION , i/WORLD_DIMENSION );
            break;
        }
    }
}

void mainLoop(){
    sf::RenderWindow WINDOW;
    
    WINDOW.create( sf::VideoMode( SCREEN_WIDTH, SCREEN_HEIGHT, 32), "PredatorPrey" );
    WINDOW.setFramerateLimit( 30 );
    CudaWorld::init();
    while (WINDOW.isOpen()) {
        
        sf::Event event;    
        while (WINDOW.pollEvent(event)) {
            if (event.type == sf::Event::Closed) { WINDOW.close(); }
        }
        // Process
        CudaWorld::process();

        // Draw
        WINDOW.clear( sf::Color(105,105,105) );
        
        render( CudaWorld::getWorld() , &WINDOW);

        WINDOW.display();
    }
     CudaWorld::end();
}

void testCudaWorld(){

    float time;
    hipEvent_t start, stop;
    CudaWorld::init();

    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    for(int i = 0; i < 5000; i++){
        CudaWorld::process();
    }

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("Time to generate:  %3.1f ms \n", time);
}


void testCudaWorldShared(){

    float time;
    hipEvent_t start, stop;
    CudaWorldShared::init();

    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    for(int i = 0; i < 5000; i++){
        CudaWorldShared::process();
    }

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("Time to generate:  %3.1f ms \n", time);
}


void testCPUWorld(){

    CPUWorld::init();
    clock_t t;
    t = clock();

    for(int i = 0; i < 5000; i++){
        CPUWorld::process();
    }

    
    t = clock() - t;
    double time_taken = ((double)t); // in seconds

    
    printf("CPU %f [ms] \n", time_taken);
    CPUWorld::end();
}


void test(int version){
    switch (version){
    case 0:
        testCudaWorld();
        break;
    case 1:
        testCudaWorldShared();
        break;
    case 2:
        testCPUWorld();
        break;

    }

}



int main(){
    std::cout<< "Running..." << std::endl;
    //mainLoop();
    test(0);

    return 0;
}